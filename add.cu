#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

// using namespace std;

#define N (1024*256)    // 每次处理的数据量
#define SIZE (N*20)     // 数据总量

// 向量加法核函数
__global__ void add(int *a, int *b, int *c)
{
    int i = blockIdx.x * blockDim.x = threadIdx.x;
    if(i < N)
        c[i] = a[i] + b[i];
}

int main()
{
    hipDeviceProp_t mprop;
    hipGetDeviceProperties(&mprop, 0);
    if(!mprop.deviceOverlap)
    {
        printf("Device not support overlaps, so stream is invalid!\n");
        return 0;
    }

    // 创建计时事件
    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);
    float elapsedTime;

    // 创建流
    hipStream_t stream0, stream1;
    hipStreamCreate(&stream0);
    hipStreamCreate(&stream1);

    // 开辟主机页锁定内存，并随机初始化数据
    int *host_a, *host_b, *host_c;
    hipHostAlloc((void**)&host_a, SIZE*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_b, SIZE*sizeof(int), hipHostMallocDefault);
    hipHostAlloc((void**)&host_c, SIZE*sizeof(int), hipHostMallocDefault);
    for(size_t i = 0; i < SIZE; i++) 
    {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    // 声明并开辟相关变量内存
    int *dev_a0, *dev_b0, *dev_c0;  // 用于流0的数据
    int *dev_a1, *dev_b1, *dev_c1;  // 用于流1的数据
    hipMalloc((void**)&dev_a0, N*sizeof(int));
    hipMalloc((void**)&dev_b0, N*sizeof(int));
    hipMalloc((void**)&dev_c0, N*sizeof(int));
    hipMalloc((void**)&dev_a1, N*sizeof(int));
    hipMalloc((void**)&dev_b1, N*sizeof(int));
    hipMalloc((void**)&dev_c1, N*sizeof(int));

    hipEventRecord(start, 0);
    for(size_t i = 0; i < SIZE; i += 2*N)
    {
        // 复制流0数据a
        hipMemcpyAsync(dev_a0, host_a+i, N*sizeof(int), hipMemcpyHostToDevice, stream0);
        // 复制流1数据a
        hipMemcpyAsync(dev_a1, host_a+i+N, N*sizeof(int), hipMemcpyHostToDevice, stream1);
        // 复制流0数据b
        hipMemcpyAsync(dev_b0, host_b+i, N*sizeof(int), hipMemcpyHostToDevice, stream0);
        // 复制流1数据b
        hipMemcpyAsync(dev_b0, host_b+i+N, N*sizeof(int), hipMemcpyHostToDevice, stream1);
        // 执行流0核函数
        add<<<N/256,256,stream0>>>(dev_a0,dev_b0,dev_c0);
        // 执行流1核函数
        add<<<N/256,256,stream1>>>(dev_a1,dev_b1,dev_c1);
        // 复制流0数据c
        hipMemcpyAsync(host_c+i*N, dev_c0, N*sizeof(int), hipMemcpyDeviceToHost, stream0);
        // 复制流1数据c
        hipMemcpyAsync(host_c+i*N+N, dev_c1, N*sizeof(int), hipMemcpyDeviceToHost, stream1);
    }

    // 流同步
    hipStreamSynchronize(stream0);
    hipStreamSynchronize(stream1);

    // 处理计时
    hipEventSynchronize(stop);
    hipEventRecord(stop, 0);
    hipEventElapsedTime(&elapsedTime, start, stop);
    // cout << "GPU Time: " << elapsedTime << "ms" << endl;

    // 销毁所有开辟的内存
    hipHostFree(host_a);
    hipHostFree(host_b);
    hipHostFree(host_c);
    hipFree(dev_a0); hipFree(dev_b0); hipFree(dev_c0);
    hipFree(dev_a1); hipFree(dev_b1); hipFree(dev_c1);

    // 销毁流以及计时事件
    hipStreamDestroy(stream0);
    hipStreamDestroy(stream1);
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}